#include "hip/hip_runtime.h"
#define __CUDA_ARCH__ 200

#include "thrust_funcs_double.h"


#define NO_ATOMICADD

__device__ double renom_atomicAdd(double* address, double val)
{

    unsigned long long int* address_as_ull = (unsigned long long int*)address;

    unsigned long long int old = *address_as_ull, assumed;

    do{ assumed = old;
        old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val +__longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

#include "thrust_funcs.inl"
